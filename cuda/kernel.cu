#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../utils/options.h"
#include "../utils/timing.h"
#include "kernel/test.cu"

extern "C" {
#include "kernel.h"
}

void calculate_dimensions(size_t[3], size_t[3], int, int);

/*
__global__ void main_kernel(struct partecl_input* inputs, struct partecl_result* results)
{
  printf("Hello %d!\n", calculateGlobalThreadId());
}
*/

extern "C" void exec_kernel(
    int do_compare_results,
    int num_runs,
    int do_time,
    int ldim0,
    int num_chunks,
    int num_test_cases,
    struct partecl_input *inputs,
    struct partecl_result *results,
    size_t size_inputs,
    size_t size_results)
{
  int chunksize = num_test_cases/num_chunks;

  //clalculate dimensions
  size_t gdim[3], ldim[3]; //assuming three dimensions
  calculate_dimensions(/*&device, */gdim, ldim, chunksize, ldim0);
  printf("LDIM = %zd\n", ldim[0]);

  if(do_time)
  {
    printf("Number of test cases: %d\n", num_test_cases);
    printf("Time in ms\n");
    printf("trans-inputs trans-results exec-kernel time-total \n");
  }

  for(int i=0; i < num_runs; i++)
  {
    //timing variables
    float trans_inputs = 0.0;
    float trans_results = 0.0;
    float time_gpu = 0.0;
    double end_to_end = 0.0;
    struct timespec ete_start, ete_end;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //allocate device memory
    partecl_input *d_inputs;
    hipMalloc(&d_inputs, size_inputs);

    partecl_result *d_results;
    hipMalloc(&d_results, size_results);

    get_timestamp(&ete_start);

    for(int j = 0; j < num_chunks; j++)
    {
      //transfer input to device
      hipEventRecord(start);
      hipMemcpy(d_inputs, inputs, size_inputs, hipMemcpyHostToDevice);
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&trans_inputs, start, stop);

      //launch kernel
      dim3 grid(gdim[0]/ldim[0], gdim[1]/ldim[1], gdim[2]/ldim[2]);
      dim3 block(ldim[0], ldim[1], ldim[2]);
      hipEventRecord(start);
      main_kernel<<<grid, block>>>(d_inputs, d_results);
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&time_gpu, start, stop);

      //transfer results back
      hipEventRecord(start);
      hipMemcpy(results, d_results, size_results, hipMemcpyDeviceToHost);
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&trans_results, start, stop);
   }

   get_timestamp(&ete_end);

   //free memory buffers
   hipFree(d_inputs);
   hipFree(d_results);

   end_to_end = timestamp_diff_in_seconds(ete_start, ete_end) * 1000; //in ms
   if(do_time)
    printf("%f %f %f %f \n", trans_inputs, trans_results, time_gpu, end_to_end);
  }
}

void calculate_dimensions(/*cl_device_id *device, */size_t gdim[3], size_t ldim[3], int num_test_cases, int ldimsupplied)
{
  //find out maximum dimensions for device
  struct hipDeviceProp_t props;
  hipGetDeviceProperties(&props, 0);
  size_t dims[3];
  dims[0] = props.maxThreadsDim[0];
  dims[1] = props.maxThreadsDim[1];
  dims[2] = props.maxThreadsDim[2];

  //calculate local dimension
  int ldim0 = num_test_cases;

  if(ldimsupplied != LDIM)
  {
    //use the given dimension
    ldim0 = ldimsupplied;
  }
  else
  {
    //calculate a dimension
    int div = num_test_cases/ dims[0]; //maximum size per work-group
    if(div > 0)
      ldim0 = num_test_cases/ (div+1);

    //ensure that the dimensions will be properly distributed across 
    while((num_test_cases / ldim0) * ldim0 != num_test_cases)
    {
      div++;
      if(div > 0)
        ldim0 = num_test_cases / div;
    }
  }

  gdim[0] = num_test_cases;
  gdim[1] = 1;
  gdim[2] = 1;
  ldim[0] = ldim0;
  ldim[1] = 1;
  ldim[2] = 1;
}

